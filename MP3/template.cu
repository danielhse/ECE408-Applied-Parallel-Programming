#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 1

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  __shared__ float tile1[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tile2[TILE_WIDTH][TILE_WIDTH];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row_o = blockIdx.y * TILE_WIDTH + ty;
  int col_o = blockIdx.x * TILE_WIDTH + tx;

  float Pvalue = 0.0;
  for (int k = 0; k < (numAColumns/TILE_WIDTH); k++) {
  // int row_i = k * TILE_WIDTH + ty;
  // int col_i = k * TILE_WIDTH + tx;
    if (row_o >= 0) {
      tile1[ty][tx] = A[row_o * numAColumns + k * TILE_WIDTH + tx];
    }
    if (col_o >= 0) {
      tile2[ty][tx] = B[(k * TILE_WIDTH + ty) * numBColumns + col_o];
    }
    __syncthreads();
    
    if (ty < TILE_WIDTH && tx < TILE_WIDTH) {
      for (int j = 0; j < TILE_WIDTH; ++j) {
        Pvalue += tile1[ty][j] * tile2[j][tx];
        __syncthreads();
      }
    }
  }
 C[row_o * numCColumns + col_o] = Pvalue;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)


  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**)&deviceA, (numARows * numAColumns * sizeof(float)));
  hipMalloc((void**)&deviceB, (numBRows * numBColumns * sizeof(float)));
  hipMalloc((void**)&deviceC, (numCRows * numCColumns * sizeof(float)));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, (numARows * numAColumns * sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, (numBRows * numBColumns * sizeof(float)), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((1.0 * numCColumns)/TILE_WIDTH), ceil((1.0 * numCRows)/TILE_WIDTH), 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, (numCRows * numCColumns * sizeof(float)), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
